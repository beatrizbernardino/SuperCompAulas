#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
//INSIRA AS IMPORTACOES NECESSARIAS DA THRUST
#include <cmath>

#include <random>

#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>


 using namespace std::chrono;

 void reportTime(const char* msg, steady_clock::duration span) {
     auto ms = duration_cast<milliseconds>(span);
     std::cout << msg << " - levou - " <<
      ms.count() << " milisegundos" << std::endl;
 }

 // CRIE UMA FUNCTOR PARA CALCULAR A SQUARE

 struct square{
     __host__ __device__
     float operator()(float x)
     {
        return x*x;
     }
 };



 // IMPLEMENTE O CALCULO DA MAGNITUDE COM THRUST
 float magnitude(thrust::device_vector<float> X) {
     float result;

     result= sqrt(thrust::transform_reduce(X.begin(), X.end(),square(), 0.0f,  thrust::plus<float>()));

     return result;
 }

 int main(int argc, char** argv) {
     if (argc != 2) {
         std::cerr << argv[0] << ": numero invalido de argumentos\n"; 
         std::cerr << "uso: " << argv[0] << "  tamanho do vetor\n"; 
         return 1;
     }
     int n = std::atoi(argv[1]); //numero de elementos
     steady_clock::time_point ts, te;

     // Faça um  vector em thrust 

     thrust::device_vector<float> X(n);




     // inicilize o  vector
     ts = steady_clock::now();

     std::generate(X.begin(),X.end(), std::rand);


     te = steady_clock::now();
     reportTime("Inicializacao", te - ts);

     // Calcule a magnitude do vetor
     ts = steady_clock::now();
     float len = magnitude(X);
     te = steady_clock::now();
     reportTime("Tempo para calculo", te - ts);


     std::cout << std::fixed << std::setprecision(4);
     std::cout << "Magnitude : " << len << std::endl;
 }
