#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include<vector>
#include<thrust/sequence.h>
#include<thrust/device_vector.h>
#include<thrust/host_vector.h>
#include<thrust/fill.h>
#include<thrust/transform.h>
#include<thrust/functional.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

using namespace std;
int main(){
    
    float x;
    thrust::host_vector<double> vec_cpu;
    thrust::device_vector<double> vec_gpu;

  int N=0;
    while (cin >> x) {
       vec_cpu.push_back(x);
       N++;
    }

    vec_gpu=vec_cpu;
    

    float mean_ten = thrust::reduce(vec_gpu.begin(), vec_gpu.end()) / N;

    float mean_one = thrust::reduce(vec_gpu.begin(), vec_gpu.begin()+365)/ 365;

   thrust::detail::normal_iterator<thrust::device_ptr<double>> max = thrust::max_element(vec_gpu.begin(), vec_gpu.begin()+365);
   thrust::detail::normal_iterator<thrust::device_ptr<double>> min = thrust::min_element(vec_gpu.begin(), vec_gpu.begin()+365);

    cout<< " 10 anos: "<< mean_ten<< endl;
    cout<< " 1 ano: "<< mean_one << endl;
    cout<< " max: "<< *max<< endl;
    cout<< " min: "<< *min<< endl;




    return 0;
}
