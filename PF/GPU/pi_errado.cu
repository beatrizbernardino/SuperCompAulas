#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include <math.h> 
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <algorithm>
#include <random>
#include<vector>
#include<thrust/sequence.h>
#include<thrust/device_vector.h>
#include<thrust/host_vector.h>
#include<thrust/fill.h>
#include<thrust/transform.h>
#include<thrust/functional.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include <thrust/count.h>
#include <thrust/random.h>
#include <thrust/iterator/counting_iterator.h>

using namespace std;

struct rng_gpu {
  
    __device__ __host__
    int operator() ( int i) {
        
      thrust::default_random_engine eng(i*18998);

      thrust::uniform_real_distribution<float> d(0, 1);

      float x= d(eng);
      float y=d(eng);

      if(pow(x,2)+pow(y,2)<=1){
          return 1;
      }else{
          
        return 0;
      }
     
    }
};



 int main(){

    int N=100000000;
    thrust::device_vector<int> d_random(N);

    int sum=thrust::transform_reduce(thrust::make_counting_iterator<int>(0),
                  thrust::make_counting_iterator<int>(N),
                 rng_gpu(), 0.0, thrust::plus<float>());
          
            
  

    float res=  4.0* (sum/ N);

    cout<< fixed<<setprecision(15) <<res<<endl;
            

  


  

     return 0;
     
 }